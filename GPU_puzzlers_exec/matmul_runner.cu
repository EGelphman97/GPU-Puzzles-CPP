#include <iostream>
#include <cassert>
#include <hip/hip_runtime.h>

const int TPB = 3;

extern __global__ void Matmul(float* A, float* B, float* C, int size);

void runKernel() {
    const int size = 2;
    float A[size][size], B[size][size], C[size][size];

    for (int i = 0; i < size; i++) {
        for (int j = 0; j < size; j++) {
        A[i][j] = i * j;
        B[i][j] = i + j;
        }
    }

    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, (size * size) * sizeof(float));
    hipMalloc(&d_B, (size * size) * sizeof(float));
    hipMalloc(&d_C, (size * size) * sizeof(float));

    hipMemcpy(d_A, A, (size * size) * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, (size * size) * sizeof(float), hipMemcpyHostToDevice);

    int BpG = (size + TPB - 1) / TPB;
    dim3 blocksPerGrid(BpG, BpG);
    dim3 threadsPerBlock(TPB, TPB);
    int sharedMemSize = 2 * (TPB * TPB) * sizeof(float);

    Matmul<<<blocksPerGrid, threadsPerBlock, sharedMemSize>>>(d_A, d_B, d_C, size);

    hipMemcpy(C, d_C, (size * size) * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    assert(C[0][0] == 0);
    assert(C[0][1] == 0);
    assert(C[1][0] == 1);
    assert(C[1][1] == 2);

    std::cout << "Matrix multiplication successful!" << std::endl;
}

int main() {
    runKernel();
    return 0;
}