#include <iostream>
#include <cassert>
#include <hip/hip_runtime.h>

extern __global__ void Blocks(float* A, float* C, float size);

void runKernel() {
    const int size = 5;
    float A[size], C[size];

    for (int i = 0; i < size; i++) {
        A[i] = static_cast<float>(i);
    }

    float *d_A, *d_C;

    hipMalloc(&d_A, size * sizeof(float));
    hipMalloc(&d_C, size * sizeof(float));

    hipMemcpy(d_A, A, size * sizeof(float), hipMemcpyHostToDevice);

    int threadsPerBlock = size - 1;
    int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;

    Blocks<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_C, size);

    hipMemcpy(C, d_C, size * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_C);

    for (int i = 0; i < size; i++) {
        assert(C[i] == A[i] + 10);
    }

    std::cout << "Blocks successful!" << std::endl;
}

int main() {
    runKernel();
    return 0;
}